// Berat Postalcioglu
/* OUTPUT
	Enter number of children: 5
	15
	3
	14
	13
	12
	cpu average: 11.4
	gpu average: 11.4
	Average is calculated correctly for 5 children as 11.4
*/
#include "hip/hip_runtime.h"

#include <cstdlib>   
#include <ctime>  
#include <iostream>
#include <cmath>

using namespace std;

void init_arr(int** arr, int size)
{

	for (int i = 0; i < size; i++)
	{
		*(*arr + i) = rand() % 16 + 1;
		cout << *(*arr + i) << endl;
	}
}

double calculate_average_oncpu(int** arr, int size)
{
	int total = 0;
	for (int i = 0; i < size; i++)
	{
		total += *(*arr + i);
	}
	return (double)total / size;
}

__global__ void calculate_average_ongpu(int *arr, int *size, double *res)
{
	int total = 0;
	for (int i = 0; i < *size; i++)
	{
		total += arr[i];
	}
	*res = ((double)total / *size);
}

int main()
{
	srand(time(NULL));

	int *arr;
	int size;
	cout << "Enter number of children: ";
	cin >> size;
	arr = new int[size];
	init_arr(&arr, size);
	double cpu_av = calculate_average_oncpu(&arr, size);
	cout << "cpu average: " << cpu_av << endl;

	// cuda
	int *gpu_arr, *gpu_size;
	double *gpu_av;
	hipMalloc((void**)&gpu_arr, size * sizeof(int));
	hipMemcpy((void*)gpu_arr, (const void*)arr, size * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&gpu_size, sizeof(int));
	hipMemcpy((void*)gpu_size, (const void*)&size, sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&gpu_av, sizeof(double));
	calculate_average_ongpu<<<1, 1>>> (gpu_arr, gpu_size, gpu_av);
	
	double *_gpu_av = new double;
	hipMemcpy((void*)_gpu_av, (const void*)gpu_av, sizeof(double), hipMemcpyDeviceToHost);
	cout << "gpu average: " << *_gpu_av << endl;

	if (*_gpu_av == cpu_av)
	{
		cout << "Average is calculated correctly for " << size << " children as " << cpu_av << endl;
	}
}
